#include "Tensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <hipDNN.h>
#include <numeric>
#include <sstream>
#include <iomanip>
#include <iostream>

#define RESET "\033[0m"
#define WHITE "\033[37m"
#define BLUE "\033[34m"
#define RED "\033[31m"
#define GREEN "\033[32m"
#define YELLOW "\033[33m"
#define MAGENTA "\033[35m"
#define CYAN "\033[36m"
#define BRIGHT_WHITE "\033[97m"
#define BRIGHT_BLUE "\033[94m"
#define BRIGHT_RED "\033[91m"
#define BRIGHT_GREEN "\033[92m"
#define BRIGHT_YELLOW "\033[93m"
#define BRIGHT_MAGENTA "\033[95m"
#define BRIGHT_CYAN "\033[96m"

namespace ryupy
{
    Tensor::Tensor(std::vector<int> shape) : Tensor()
    {
        if (std::any_of(shape.begin(), shape.end(), [](int d)
                        { return d <= 0; }))
        {
            throw std::invalid_argument("All dimensions must be positive");
        }

        size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>()) * sizeof(float);
        this->size = size;
        this->shape = shape;

        hipMalloc(&d_data, size);
        hipdnnCreateTensorDescriptor(&tensor_desc);

        // For cuDNN, convert dimensions to NCHW format if needed
        int nbDims = shape.size();
        while (nbDims < 4)
        {
            shape.insert(shape.begin(), 1);
            nbDims++;
        }

        std::vector<int> stride(nbDims);
        stride[0] = 1;
        for (int i = 1; i < nbDims; i++)
        {
            stride[i] = stride[i - 1] * shape[i - 1];
        }

        hipdnnStatus_t status = hipdnnSetTensorNdDescriptor(
            tensor_desc,
            HIPDNN_DATA_FLOAT,
            nbDims,
            shape.data(),
            stride.data());

        if (status != HIPDNN_STATUS_SUCCESS)
        {
            throw std::runtime_error(std::string("CUDNN error: ") + hipdnnGetErrorString(status));
        }
    }

    Tensor::~Tensor()
    {
        if (d_data != nullptr)
        {
            hipFree(d_data);
            d_data = nullptr;
        }
        if (tensor_desc != nullptr)
        {
            hipdnnDestroyTensorDescriptor(tensor_desc);
            tensor_desc = nullptr;
        }
    }
}
