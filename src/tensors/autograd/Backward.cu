#include "../Tensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <optional>
#include <iostream>

namespace ryupy
{
    void Tensor::backward(const Tensor *gradient)
    {
        if (!requires_grad)
        {
            throw std::runtime_error("Tensor does not require gradients");
        }
        if (!backward_fn)
        {
            throw std::runtime_error("Tensor has no backward function");
        }

        if (!grad)
        {
            if (gradient != nullptr)
            {
                grad = std::make_shared<Tensor>(*gradient);
            }
            else
            {
                grad = Tensor::ones(shape);
            }
        }

        if (is_leaf)
        {
            return;
        }

        backward_fn();

        for (const auto &prev : prev)
        {
            if (prev->is_leaf == false)
            {
                prev->backward();
            }
        }
    }
}