#include "Tensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <hipDNN.h>
#include <numeric>
#include <sstream>
#include <iomanip>
#include <iostream>

#define RESET "\033[0m"
#define WHITE "\033[37m"
#define BLUE "\033[34m"
#define RED "\033[31m"
#define GREEN "\033[32m"
#define YELLOW "\033[33m"
#define MAGENTA "\033[35m"
#define CYAN "\033[36m"
#define BRIGHT_WHITE "\033[97m"
#define BRIGHT_BLUE "\033[94m"
#define BRIGHT_RED "\033[91m"
#define BRIGHT_GREEN "\033[92m"
#define BRIGHT_YELLOW "\033[93m"
#define BRIGHT_MAGENTA "\033[95m"
#define BRIGHT_CYAN "\033[96m"

namespace ryupy
{
    py::object Tensor::getFlattenedData() const
    {
        std::vector<float> hostData(size / sizeof(float));
        hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
        return py::cast(hostData);
    }

    py::object Tensor::getData() const
    {
        std::vector<float> hostData(size / sizeof(float));
        hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
        return reshapeData(hostData, shape);
    }

    const std::vector<int> Tensor::getShape() const
    {
        return shape;
    }

    py::object Tensor::getItem(int index)
    {
        if (index < 0 || index >= shape[0])
        {
            throw std::out_of_range("Index out of bounds");
        }

        // For 1D tensors, return a single value
        if (shape.size() == 1)
        {
            float value;
            hipMemcpy(&value, d_data + index, sizeof(float), hipMemcpyDeviceToHost);
            return py::cast(value);
        }

        // For multi-dimensional tensors, return a slice
        std::vector<int> newShape(shape.begin() + 1, shape.end());
        int sliceSize = std::accumulate(newShape.begin(), newShape.end(), 1, std::multiplies<int>());

        auto slice = std::make_shared<Tensor>(newShape);
        // Store the parent tensor and index for later use in setItem
        slice->parent = shared_from_this();
        slice->parent_index = index;

        hipMemcpy(slice->d_data,
                   d_data + (index * sliceSize),
                   sliceSize * sizeof(float),
                   hipMemcpyDeviceToDevice);

        return py::cast(slice);
    }

    void Tensor::setItem(int index, const py::object &value)
    {
        if (index < 0 || index >= shape[0])
        {
            throw std::out_of_range("Index out of bounds");
        }

        // Get the base tensor and accumulated index
        float *target_data;
        int total_offset;

        if (parent)
        {
            // This is a slice, so calculate offset into parent tensor
            int parent_stride = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
            total_offset = (parent_index * parent_stride) + index;
            target_data = parent->d_data;
        }
        else
        {
            // This is the root tensor
            total_offset = index;
            target_data = d_data;
        }

        // Set the single value
        float newValue = value.cast<float>();
        hipMemcpy(target_data + total_offset,
                   &newValue,
                   sizeof(float),
                   hipMemcpyHostToDevice);
    }

    std::string Tensor::repr() const
    {
        std::stringstream ss;
        ss << RED << "[";

        for (size_t i = 0; i < shape.size(); i++)
        {
            ss << shape[i];
            if (i < shape.size() - 1)
                ss << ", ";
        }

        ss << "]";
        ss << std::endl;
        ss << MAGENTA;

        int total_elements = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
        std::vector<float> h_data(total_elements);
        hipMemcpy(h_data.data(), d_data, total_elements * sizeof(float), hipMemcpyDeviceToHost);

        if (shape.size() == 1)
        {
            ss << "[";
            const int max_preview = 6;
            if (total_elements <= max_preview)
            {
                for (int i = 0; i < total_elements; i++)
                {
                    ss << std::fixed << std::setprecision(4) << h_data[i];
                    if (i < total_elements - 1)
                        ss << ", ";
                }
            }
            else
            {
                for (int i = 0; i < 3; i++)
                {
                    ss << std::fixed << std::setprecision(4) << h_data[i] << ", ";
                }
                ss << "..., ";
                for (int i = total_elements - 3; i < total_elements; i++)
                {
                    ss << std::fixed << std::setprecision(4) << h_data[i];
                    if (i < total_elements - 1)
                        ss << ", ";
                }
            }
            ss << "]";
        }

        else if (shape.size() == 2)
        {
            ss << "[\n";
            int rows = shape[0];
            int cols = shape[1];
            for (int i = 0; i < rows; i++)
            {
                ss << " [";
                for (int j = 0; j < cols; j++)
                {
                    ss << std::fixed << std::setprecision(4) << h_data[i * cols + j];
                    if (j < cols - 1)
                        ss << ", ";
                }
                ss << "]";
                if (i < rows - 1)
                    ss << ",\n";
            }
            ss << "\n]";
        }

        else if (shape.size() == 3)
        {
            ss << "[\n";
            int dim1 = shape[0];
            int dim2 = shape[1];
            int dim3 = shape[2];
            for (int i = 0; i < dim1; i++)
            {
                ss << " [\n";
                for (int j = 0; j < dim2; j++)
                {
                    ss << "  [";
                    for (int k = 0; k < dim3; k++)
                    {
                        ss << std::fixed << std::setprecision(4)
                           << h_data[i * dim2 * dim3 + j * dim3 + k];
                        if (k < dim3 - 1)
                            ss << ", ";
                    }
                    ss << "]";
                    if (j < dim2 - 1)
                        ss << ",\n";
                }
                ss << "\n ]";
                if (i < dim1 - 1)
                    ss << ",\n";
            }
            ss << "\n]";
        }

        else
        {
            ss << "<tensor of size ";
            for (size_t i = 0; i < shape.size(); i++)
            {
                ss << shape[i];
                if (i < shape.size() - 1)
                    ss << "×";
            }
            ss << ">";
        }

        ss << RESET;

        return ss.str();
    }
}
