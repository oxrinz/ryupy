#include "../Tensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include "kernels/Kernels.h"

namespace ryupy
{
    std::shared_ptr<Tensor> Tensor::operator==(Tensor &other)
    {
        return handleOperator(other, equalityKernel);
    }

    std::shared_ptr<Tensor> Tensor::operator!=(Tensor &other)
    {
        return handleOperator(other, inequalityKernel);
    }

    std::shared_ptr<Tensor> Tensor::operator<(Tensor &other)
    {
        return handleOperator(other, lessThanKernel);
    }

    std::shared_ptr<Tensor> Tensor::operator<=(Tensor &other)
    {
        return handleOperator(other, lessThanOrEqualKernel);
    }

    std::shared_ptr<Tensor> Tensor::operator>(Tensor &other)
    {
        return handleOperator(other, greaterThanKernel);
    }

    std::shared_ptr<Tensor> Tensor::operator>=(Tensor &other)
    {
        return handleOperator(other, greaterThanOrEqualKernel);
    }
}
