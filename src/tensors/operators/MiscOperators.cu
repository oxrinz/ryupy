#include "hip/hip_runtime.h"
#include "../Tensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include "kernels/Kernels.h"
#include <hipblas.h>
#include <memory>
#include <numeric>
#include <iostream>

namespace ryupy
{

    std::shared_ptr<Tensor> Tensor::pow(Tensor &other)
    {
        return handleOperator(other, powerKernel, &Tensor::powerBackward);
    }

    std::shared_ptr<Tensor> Tensor::ipow(Tensor &other)
    {
        return handleInPlaceOperator(other, powerKernel);
    }

    std::shared_ptr<Tensor> Tensor::matmul(Tensor &other)
    {
        const float alpha = 1.0f;
        const float beta = 0.0f;
        hipblasHandle_t handle;

        if (hipblasCreate(&handle) != HIPBLAS_STATUS_SUCCESS)
        {
            throw std::runtime_error("Failed to create cuBLAS handle");
        }

        std::unique_ptr<std::remove_pointer<hipblasHandle_t>::type, decltype(&hipblasDestroy)>
            handle_guard(handle, hipblasDestroy);

        if (shape.size() == 1 && other.shape.size() == 2)
        {
            if (shape[0] != other.shape[0])
            {
                throw std::invalid_argument("Vector dimension must match matrix's first dimension.");
            }

            std::vector<int> result_shape = {other.shape[1]};
            auto result = std::make_shared<Tensor>(other.shape[1] * sizeof(float), result_shape);

            if (requires_grad || other.requires_grad)
            {
                result->requires_grad = true;
                result->is_leaf = false;
                result->prev.push_back(shared_from_this());
                result->prev.push_back(other.shared_from_this());
                result->backward_fn = [result]()
                { result.get()->matmulBackward(); };
            }

            if (hipblasSgemv(handle, HIPBLAS_OP_T,
                            other.shape[0], other.shape[1],
                            &alpha,
                            other.d_data, other.shape[0],
                            d_data, 1,
                            &beta,
                            result->d_data, 1) != HIPBLAS_STATUS_SUCCESS)
            {
                throw std::runtime_error("hipblasSgemv failed in vector-matrix multiplication");
            }
            return result;
        }

        if (shape.size() == 2 && other.shape.size() == 1)
        {
            if (shape[1] != other.shape[0])
            {
                throw std::invalid_argument("Matrix second dimension must match vector dimension.");
            }

            std::vector<int> result_shape = {shape[0]};
            auto result = std::make_shared<Tensor>(shape[0] * sizeof(float), result_shape);

            if (requires_grad || other.requires_grad)
            {
                result->requires_grad = true;
                result->is_leaf = false;
                result->prev.push_back(shared_from_this());
                result->prev.push_back(other.shared_from_this());
                result->backward_fn = [result]()
                { result.get()->matmulBackward(); };
            }

            if (hipblasSgemv(handle, HIPBLAS_OP_N,
                            shape[0], shape[1],
                            &alpha,
                            d_data, shape[1],
                            other.d_data, 1,
                            &beta,
                            result->d_data, 1) != HIPBLAS_STATUS_SUCCESS)
            {
                throw std::runtime_error("hipblasSgemv failed in matrix-vector multiplication");
            }
            return result;
        }

        if (shape.size() >= 2 && other.shape.size() >= 2)
        {
            if (shape.size() != other.shape.size())
            {
                throw std::invalid_argument("Input tensors must have the same number of dimensions.");
            }

            int batchSize = 1;
            int batchDim = shape.size() - 2;

            for (int i = 0; i < batchDim; ++i)
            {
                if (shape[i] != other.shape[i])
                {
                    throw std::invalid_argument("Batch dimensions must match for matrix multiplication.");
                }
                batchSize *= shape[i];
            }

            int m = shape[shape.size() - 2];
            int k = shape[shape.size() - 1];
            int k_other = other.shape[other.shape.size() - 2];
            int n = other.shape[other.shape.size() - 1];

            if (k != k_other)
            {
                throw std::invalid_argument("Inner dimensions must match for matrix multiplication.");
            }

            std::vector<int> result_shape(shape.begin(), shape.begin() + batchDim);
            result_shape.push_back(m);
            result_shape.push_back(n);

            auto result = std::make_shared<Tensor>(batchSize * m * n * sizeof(float), result_shape);

            if (requires_grad || other.requires_grad)
            {
                result->requires_grad = true;
                result->is_leaf = false;
                result->prev.push_back(shared_from_this());
                result->prev.push_back(other.shared_from_this());
                result->backward_fn = [result]()
                { result.get()->matmulBackward(); };
            }

            std::cout << "m: " << m << " n: " << n << " k: " << k << " batchSize: " << batchSize << std::endl;

            if (hipblasSgemmStridedBatched(handle,
                                          HIPBLAS_OP_N, HIPBLAS_OP_N,
                                          n, m, k,
                                          &alpha,
                                          other.d_data, n, k * n, // Matrix B
                                          d_data, k, m * k,       // Matrix A
                                          &beta,
                                          result->d_data, n, m * n,
                                          batchSize) != HIPBLAS_STATUS_SUCCESS)
            {
                throw std::runtime_error("hipblasSgemmStridedBatched failed");
            }
            return result;
        }

        throw std::invalid_argument("Invalid tensor dimensions for matrix multiplication");
    }
}