#include "../../../tensors/Tensor.h"
#include "LinearLayer.h"
#include <hip/hip_runtime.h>
#include <iostream>

namespace ryupy
{
    namespace nn
    {
        LinearLayer::LinearLayer(int in_features, int out_features, InitType init_type)
        {
            std::vector<int> weight_shape = {out_features, in_features};

            switch (init_type)
            {
            case InitType::XAVIER_UNIFORM:
                weight = Tensor::xavier_uniform(weight_shape);
                break;
            case InitType::XAVIER_NORMAL:
                weight = Tensor::xavier_normal(weight_shape);
                break;
            case InitType::KAIMING_UNIFORM:
                weight = Tensor::kaiming_uniform(weight_shape);
                break;
            case InitType::KAIMING_NORMAL:
                weight = Tensor::kaiming_normal(weight_shape);
                break;
            }

            weight->requires_grad = true;

            std::vector<int> bias_shape = {out_features};
            bias = Tensor::zeros(bias_shape);

            bias->requires_grad = true;
        }

        std::shared_ptr<Tensor> LinearLayer::forward(Tensor &tensor)
        {
            return tensor.matmul(*weight->transpose(0, 1))->operator+(*bias);
        }

        std::shared_ptr<LinearLayer> LinearLayer::create(int in_features, int out_features, InitType init_type)
        {
            return std::make_shared<LinearLayer>(in_features, out_features, init_type);
        }
    }
}