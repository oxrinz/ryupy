#include "../tensors/Tensor.h"
#include "LinearLayer.h"
#include <hip/hip_runtime.h>

namespace ryupy
{
    namespace nn
    {
        LinearLayer::LinearLayer(int in_features, int out_features, InitType init_type)
        {
            std::vector<int> weight_shape = {out_features, in_features};

            switch (init_type)
            {
            case InitType::XAVIER_UNIFORM:
                weight = Tensor::xavier_uniform(weight_shape);
                break;
            case InitType::XAVIER_NORMAL:
                weight = Tensor::xavier_normal(weight_shape);
                break;
            case InitType::KAIMING_UNIFORM:
                weight = Tensor::kaiming_uniform(weight_shape);
                break;
            case InitType::KAIMING_NORMAL:
                weight = Tensor::kaiming_normal(weight_shape);
                break;
            }

            std::vector<int> bias_shape = {out_features};
            bias = Tensor::zeros(bias_shape);
        }

        std::shared_ptr<Tensor> LinearLayer::forward(Tensor &tensor)
        {
            return tensor.matmul(*weight);
        }
    }
}