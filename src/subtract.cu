#include <hip/hip_runtime.h>

__global__ void subtractKernel(int* result, int a, int b) {
    *result = a - b;
}

extern "C" __declspec(dllexport) int subtract(int a, int b) {
    int result;  
    int* d_result;  

    hipMalloc(&d_result, sizeof(int));

    subtractKernel<<<1, 1>>>(d_result, a, b);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_result);

    return result;
}
