#include <hip/hip_runtime.h>

__global__ void addKernel(int* result, int a, int b) {
    *result = a + b;
}

extern "C" __declspec(dllexport) int add(int a, int b) {
    int result;  
    int* d_result; 

    hipMalloc(&d_result, sizeof(int));

    addKernel<<<1, 1>>>(d_result, a, b);

    hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_result);

    return result;
}
