#include "../CudaTensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include "kernels/Kernels.h"

namespace ryupy
{
    namespace cuda
    {
        std::shared_ptr<CudaTensor> CudaTensor::operator==(const CudaTensor &other) const
        {
            return handleOperator(other, equalityKernel);
        }

        std::shared_ptr<CudaTensor> CudaTensor::operator!=(const CudaTensor &other) const
        {
            return handleOperator(other, inequalityKernel);
        }

        std::shared_ptr<CudaTensor> CudaTensor::operator<(const CudaTensor &other) const
        {
            return handleOperator(other, lessThanKernel);
        }

        std::shared_ptr<CudaTensor> CudaTensor::operator<=(const CudaTensor &other) const
        {
            return handleOperator(other, lessThanOrEqualKernel);
        }

        std::shared_ptr<CudaTensor> CudaTensor::operator>(const CudaTensor &other) const
        {
            return handleOperator(other, greaterThanKernel);
        }

        std::shared_ptr<CudaTensor> CudaTensor::operator>=(const CudaTensor &other) const
        {
            return handleOperator(other, greaterThanOrEqualKernel);
        }
    }
}
