#include "CudaTensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>

namespace ryupy
{
    namespace cuda
    {
        CudaTensor::CudaTensor(const py::object &py_data) : Tensor(py_data)
        {
            std::vector<float> hostData = flattenData(py_data);

            size = hostData.size() * sizeof(float);

            hipMalloc(&d_data, hostData.size() * sizeof(float));

            hipMemcpy(d_data, hostData.data(), size, hipMemcpyHostToDevice);
        }

        py::object CudaTensor::getFlattenedData() const
        {
            std::vector<float> hostData(size / sizeof(float));
            hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
            return py::cast(hostData);
        }

        py::object CudaTensor::getData() const
        {
            std::vector<float> hostData(size / sizeof(float));
            hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
            int index = 0;
            return reshapeData(hostData, shape, index);
        }

        CudaTensor::~CudaTensor()
        {
            hipFree(d_data);
        }
    }
}
