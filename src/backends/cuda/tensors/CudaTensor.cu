#include "CudaTensor.h"
#include <hip/hip_runtime.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <hipDNN.h>
#include <numeric>

namespace ryupy
{
    namespace cuda
    {
        CudaTensor::CudaTensor(const py::object &py_data) : Tensor(py_data)
        {
            std::vector<float> hostData = flattenPythonData(py_data);

            size = hostData.size() * sizeof(float);

            hipMalloc(&d_data, size);

            hipMemcpy(d_data, hostData.data(), size, hipMemcpyHostToDevice);

            hipdnnCreateTensorDescriptor(&tensor_desc);

            int nbDims = shape.size();
            std::vector<int> strideA(nbDims);

            strideA[nbDims - 1] = 1;
            for (int i = nbDims - 2; i >= 0; --i)
            {
                strideA[i] = strideA[i + 1] * shape[i + 1];
            }

            hipdnnSetTensorNdDescriptor(tensor_desc,
                                       HIPDNN_DATA_FLOAT,
                                       nbDims,
                                       shape.data(),
                                       strideA.data());
        }

        CudaTensor::CudaTensor(std::vector<int> shape) : Tensor()
        {
            size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>()) * sizeof(float);

            std::cout << "init sexing " << size << std::endl;

            this->size = size;
            this->shape = shape;

            hipMalloc(&d_data, size);

            hipdnnCreateTensorDescriptor(&tensor_desc);

            int nbDims = shape.size();
            std::vector<int> strideA(nbDims);

            strideA[nbDims - 1] = 1;
            for (int i = nbDims - 2; i >= 0; --i)
            {
                strideA[i] = strideA[i + 1] * shape[i + 1];
            }

            hipdnnSetTensorNdDescriptor(tensor_desc,
                                       HIPDNN_DATA_FLOAT,
                                       nbDims,
                                       shape.data(),
                                       strideA.data());
        }

        CudaTensor::CudaTensor(int size, std::vector<int> shape) : Tensor()
        {
            this->size = size;
            this->shape = shape;

            hipMalloc(&d_data, size);

            hipdnnCreateTensorDescriptor(&tensor_desc);

            int nbDims = shape.size();
            std::vector<int> strideA(nbDims);

            strideA[nbDims - 1] = 1;
            for (int i = nbDims - 2; i >= 0; --i)
            {
                strideA[i] = strideA[i + 1] * shape[i + 1];
            }

            hipdnnSetTensorNdDescriptor(tensor_desc,
                                       HIPDNN_DATA_FLOAT,
                                       nbDims,
                                       shape.data(),
                                       strideA.data());
        }
 
        py::object CudaTensor::getFlattenedData() const
        {
            std::vector<float> hostData(size / sizeof(float));
            hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
            return py::cast(hostData);
        }

        py::object CudaTensor::getData() const
        {
            std::cout << "sex " << size << std::endl;
            std::vector<float> hostData(size / sizeof(float));
            hipMemcpy(hostData.data(), d_data, size, hipMemcpyDeviceToHost);
            return reshapeData(hostData, shape);
        }

        CudaTensor::~CudaTensor()
        {
            hipdnnDestroyTensorDescriptor(tensor_desc);
            hipFree(d_data);
        }
    }
}
