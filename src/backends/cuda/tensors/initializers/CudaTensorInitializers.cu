#include "../CudaTensor.h"
#include "Kernels.h"
#include <numeric>
#include <hiprand.h>

namespace ryupy
{
    namespace cuda
    {
        std::shared_ptr<CudaTensor> CudaTensor::zeros(const std::vector<int> &shape)
        {
            std::shared_ptr<CudaTensor> tensor = std::make_shared<CudaTensor>(shape);

            int size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>()) * sizeof(float);

            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;

            zerosKernel<<<numBlocks, blockSize>>>(tensor->d_data, size);

            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::ones(const std::vector<int> &shape)
        {
            std::shared_ptr<CudaTensor> tensor = std::make_shared<CudaTensor>(shape);

            int size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>()) * sizeof(float);

            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;

            onesKernel<<<numBlocks, blockSize>>>(tensor->d_data, size);

            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::fill(const std::vector<int> &shape, float val)
        {
            std::shared_ptr<CudaTensor> tensor = std::make_shared<CudaTensor>(shape);

            int size = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>()) * sizeof(float);

            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;

            fillKernel<<<numBlocks, blockSize>>>(tensor->d_data, val, size);

            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::arange(float start, float stop, float step)
        {
            int size = static_cast<int>((stop - start) / step);
            std::vector<int> shape = {size};

            std::shared_ptr<CudaTensor> tensor = std::make_shared<CudaTensor>(shape);

            int blockSize = 256;
            int numBlocks = (size + blockSize - 1) / blockSize;
            arangeKernel<<<numBlocks, blockSize>>>(tensor->d_data, start, step, size);
            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::linspace(float start, float stop, int num)
        {
            std::vector<int> shape = {num};
            auto tensor = std::make_shared<CudaTensor>(shape);
            float step = (stop - start) / (num - 1);

            int blockSize = 256;
            int numBlocks = (num + blockSize - 1) / blockSize;
            linspaceKernel<<<numBlocks, blockSize>>>(tensor->d_data, start, step, num);
            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::eye(int n)
        {
            std::vector<int> shape = {n, n};
            auto tensor = std::make_shared<CudaTensor>(shape);

            int blockSize = 256;
            int numBlocks = (n * n + blockSize - 1) / blockSize;
            eyeKernel<<<numBlocks, blockSize>>>(tensor->d_data, n);
            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::random_uniform(const std::vector<int> &shape, float low, float high)
        {
            auto tensor = std::make_shared<CudaTensor>(shape);

            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

            hiprandSetPseudoRandomGeneratorSeed(gen, static_cast<unsigned long long>(time(nullptr)));

            hiprandGenerateUniform(gen, tensor->d_data, tensor->size);

            if (low != 0.0f || high != 1.0f)
            {
                float scale = high - low;
                int blockSize = 256;
                int numBlocks = (tensor->size + blockSize - 1) / blockSize;
                scaleKernel<<<numBlocks, blockSize>>>(tensor->d_data, low, scale, tensor->size);
            }
            hiprandDestroyGenerator(gen);
            return tensor;
        }

        std::shared_ptr<CudaTensor> CudaTensor::random_normal(const std::vector<int> &shape, float mean, float std)
        {
            auto tensor = std::make_shared<CudaTensor>(shape);

            hiprandGenerator_t gen;
            hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);

            hiprandSetPseudoRandomGeneratorSeed(gen, static_cast<unsigned long long>(time(nullptr)));

            hiprandGenerateNormal(gen, tensor->d_data, tensor->size, mean, std);
            hiprandDestroyGenerator(gen);
            return tensor;
        }
    }
}