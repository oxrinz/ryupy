#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/operators.h>
#include <pybind11/stl.h>
#include <pybind11/functional.h>
#include "../tensors/Tensor.h"
#include "../nn/layers/Layer.h"
#include "../nn/layers/basic/LinearLayer.h"
#include "../nn/layerbank/LayerBank.h"
#include "../nn/net/Net.h"
#include "../Ryu.h"
#include "../nn/loss/Loss.h"
#include "../nn/optim/Optim.h"

namespace py = pybind11;

PYBIND11_MODULE(_ryupy, m)
{
     m.def("ryu", &ryupy::print_ryu);

     py::class_<ryupy::Tensor, std::shared_ptr<ryupy::Tensor>>(m, "Tensor")
         .def_property_readonly("shape", &ryupy::Tensor::getShape)
         .def_property_readonly("flattenedData", &ryupy::Tensor::getFlattenedData)
         .def_property_readonly("data", &ryupy::Tensor::getData)

         .def_property("grad", [](ryupy::Tensor &t)
                       { return t.grad; }, [](ryupy::Tensor &t, std::shared_ptr<ryupy::Tensor> new_grad)
                       { t.grad = new_grad; })
         .def_readwrite("requires_grad", &ryupy::Tensor::requires_grad)
         .def("backward", &ryupy::Tensor::backward, py::arg("gradient") = nullptr)

         .def("copy", &ryupy::Tensor::copy)

         .def("__repr__", &ryupy::Tensor::repr)

         .def("__getitem__", &ryupy::Tensor::getItem)
         .def("__setitem__", &ryupy::Tensor::setItem)

         .def("__add__", &ryupy::Tensor::operator+)
         .def("__sub__", &ryupy::Tensor::operator-)
         .def("__mul__", &ryupy::Tensor::operator*)
         .def("__truediv__", &ryupy::Tensor::operator/)
         .def("__mod__", &ryupy::Tensor::operator%)

         .def("__iadd__", &ryupy::Tensor::operator+=)
         .def("__isub__", &ryupy::Tensor::operator-=)
         .def("__imul__", &ryupy::Tensor::operator*=)
         .def("__itruediv__", &ryupy::Tensor::operator/=)
         .def("__imod__", &ryupy::Tensor::operator%=)

         .def("__pow__", &ryupy::Tensor::pow)
         .def("__ipow__", &ryupy::Tensor::ipow)

         .def("__eq__", &ryupy::Tensor::operator==)
         .def("__ne__", &ryupy::Tensor::operator!=)
         .def("__lt__", &ryupy::Tensor::operator<)
         .def("__le__", &ryupy::Tensor::operator<=)
         .def("__gt__", &ryupy::Tensor::operator>)
         .def("__ge__", &ryupy::Tensor::operator>=)

         .def("__and__", &ryupy::Tensor::operator&)
         .def("__or__", &ryupy::Tensor::operator|)
         .def("__xor__", &ryupy::Tensor::operator^)
         .def("__invert__", &ryupy::Tensor::operator~)
         .def("__lshift__", &ryupy::Tensor::operator<<)
         .def("__rshift__", &ryupy::Tensor::operator>>)

         .def("__iand__", &ryupy::Tensor::operator&=)
         .def("__ior__", &ryupy::Tensor::operator|=)
         .def("__ixor__", &ryupy::Tensor::operator^=)
         .def("__ilshift__", &ryupy::Tensor::operator<<=)
         .def("__irshift__", &ryupy::Tensor::operator>>=)

         .def("sum", &ryupy::Tensor::sum)
         .def("__neg__", &ryupy::Tensor::negate)

         .def("__matmul__", &ryupy::Tensor::matmul);

     m.def("zeros", &ryupy::Tensor::zeros,
           py::arg("shape"),
           py::kw_only(),
           py::arg("grad") = false)
         .def("ones", &ryupy::Tensor::ones,
              py::arg("shape"),
              py::kw_only(),
              py::arg("grad") = false)
         .def("fill", &ryupy::Tensor::fill,
              py::arg("shape"),
              py::arg("value"),
              py::kw_only(),
              py::arg("grad") = false)
         .def("arange", &ryupy::Tensor::arange,
              py::arg("start"),
              py::arg("stop"),
              py::kw_only(),
              py::arg("step") = 1.0f,
              py::arg("grad") = false)
         .def("linspace", &ryupy::Tensor::linspace,
              py::arg("start"),
              py::arg("stop"),
              py::arg("num"),
              py::kw_only(),
              py::arg("grad") = false)
         .def("rand", &ryupy::Tensor::random_uniform,
              py::arg("shape"),
              py::kw_only(),
              py::arg("low") = 0.0f,
              py::arg("high") = 1.0f,
              py::arg("grad") = false)
         .def("randn", &ryupy::Tensor::random_normal,
              py::arg("shape"),
              py::kw_only(),
              py::arg("mean") = 0.0f,
              py::arg("std") = 1.0f,
              py::arg("grad") = false);

     auto nn = m.def_submodule("nn");

     using InitType = ryupy::nn::LinearLayer::InitType;
     py::enum_<InitType>(nn, "InitType")
         .value("XAVIER_UNIFORM", InitType::XAVIER_UNIFORM)
         .value("XAVIER_NORMAL", InitType::XAVIER_NORMAL)
         .value("KAIMING_UNIFORM", InitType::KAIMING_UNIFORM)
         .value("KAIMING_NORMAL", InitType::KAIMING_NORMAL)
         .export_values();

     py::class_<ryupy::nn::Layer, std::shared_ptr<ryupy::nn::Layer>>(nn, "Layer")
         .def("forward", &ryupy::nn::Layer::forward)
         .def("__call__", &ryupy::nn::Layer::forward);

     py::class_<ryupy::nn::LinearLayer, ryupy::nn::Layer, std::shared_ptr<ryupy::nn::LinearLayer>>(nn, "Linear")
         .def(py::init(&ryupy::nn::LinearLayer::create))
         .def_readwrite("weight", &ryupy::nn::LinearLayer::weight)
         .def_readwrite("bias", &ryupy::nn::LinearLayer::bias);

     py::class_<ryupy::nn::LayerBank, std::shared_ptr<ryupy::nn::LayerBank>>(nn, "LayerBank")
         .def(py::init(&ryupy::nn::LayerBank::create))
         .def("__setattr__", &ryupy::nn::LayerBank::setLayer)
         .def("__getattr__", &ryupy::nn::LayerBank::getLayer);

     py::class_<ryupy::nn::Net, std::shared_ptr<ryupy::nn::Net>>(nn, "Net")
         .def(py::init(&ryupy::nn::Net::create))
         .def("__call__", &ryupy::nn::Net::forward);

     auto loss = nn.def_submodule("loss");

     loss.def("mse", &ryupy::nn::loss::mse_loss);

     auto optim = nn.def_submodule("optim");

     py::class_<ryupy::nn::optim::Optimizer, std::shared_ptr<ryupy::nn::optim::Optimizer>>(optim, "Optimizer")
         .def(py::init<std::shared_ptr<ryupy::nn::LayerBank>, float>())
         .def("step", &ryupy::nn::optim::Optimizer::step);

     using SGD = ryupy::nn::optim::SGD;
     py::class_<SGD, ryupy::nn::optim::Optimizer, std::shared_ptr<SGD>>(optim, "SGD")
         .def(py::init<std::shared_ptr<ryupy::nn::Net>, float, float, float, float, bool>(),
              py::arg("layer_bank"),
              py::arg("lr") = 0.01f,
              py::arg("momentum") = 0.0f,
              py::arg("dampening") = 0.0f,
              py::arg("weight_decay") = 0.0f,
              py::arg("nesterov") = false)
         .def("step", &SGD::step);
}