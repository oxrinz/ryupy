#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/operators.h>
#include <pybind11/stl.h>
#include "../tensors/Tensor.h"
#include "../layers/LinearLayer.h"

namespace py = pybind11;

PYBIND11_MODULE(_ryupy, m)
{
    py::class_<ryupy::Tensor, std::shared_ptr<ryupy::Tensor>>(m, "Tensor")
        .def(py::init<py::object>())
        .def_property_readonly("shape", &ryupy::Tensor::getShape)
        .def_property_readonly("flattenedData", &ryupy::Tensor::getFlattenedData)
        .def_property_readonly("data", &ryupy::Tensor::getData)

        .def_property("grad", [](ryupy::Tensor &t)
                      { return t.grad; }, [](ryupy::Tensor &t, std::shared_ptr<ryupy::Tensor> new_grad)
                      { t.grad = new_grad; })
        .def_readwrite("requires_grad", &ryupy::Tensor::requires_grad)
        .def("backward", &ryupy::Tensor::backward)

        .def("__repr__", &ryupy::Tensor::repr)

        .def("__add__", &ryupy::Tensor::operator+)
        .def("__sub__", &ryupy::Tensor::operator-)
        .def("__mul__", &ryupy::Tensor::operator*)
        .def("__truediv__", &ryupy::Tensor::operator/)
        .def("__mod__", &ryupy::Tensor::operator%)

        .def("__iadd__", &ryupy::Tensor::operator+=)
        .def("__isub__", &ryupy::Tensor::operator-=)
        .def("__imul__", &ryupy::Tensor::operator*=)
        .def("__itruediv__", &ryupy::Tensor::operator/=)
        .def("__imod__", &ryupy::Tensor::operator%=)

        .def("__pow__", &ryupy::Tensor::pow)
        .def("__ipow__", &ryupy::Tensor::ipow)

        .def("__eq__", &ryupy::Tensor::operator==)
        .def("__ne__", &ryupy::Tensor::operator!=)
        .def("__lt__", &ryupy::Tensor::operator<)
        .def("__le__", &ryupy::Tensor::operator<=)
        .def("__gt__", &ryupy::Tensor::operator>)
        .def("__ge__", &ryupy::Tensor::operator>=)

        .def("__and__", &ryupy::Tensor::operator&)
        .def("__or__", &ryupy::Tensor::operator|)
        .def("__xor__", &ryupy::Tensor::operator^)
        .def("__invert__", &ryupy::Tensor::operator~)
        .def("__lshift__", &ryupy::Tensor::operator<<)
        .def("__rshift__", &ryupy::Tensor::operator>>)

        .def("__iand__", &ryupy::Tensor::operator&=)
        .def("__ior__", &ryupy::Tensor::operator|=)
        .def("__ixor__", &ryupy::Tensor::operator^=)
        .def("__ilshift__", &ryupy::Tensor::operator<<=)
        .def("__irshift__", &ryupy::Tensor::operator>>=)

        .def("__matmul__", &ryupy::Tensor::matmul);

    m.def("zeros", &ryupy::Tensor::zeros,
          py::arg("shape"),
          py::kw_only(),
          py::arg("grad") = false)
        .def("ones", &ryupy::Tensor::ones,
             py::arg("shape"),
             py::kw_only(),
             py::arg("grad") = false)
        .def("fill", &ryupy::Tensor::fill,
             py::arg("shape"),
             py::arg("value"),
             py::kw_only(),
             py::arg("grad") = false)
        .def("arange", &ryupy::Tensor::arange,
             py::arg("start"),
             py::arg("stop"),
             py::kw_only(),
             py::arg("step") = 1.0f,
             py::arg("grad") = false)
        .def("linspace", &ryupy::Tensor::linspace,
             py::arg("start"),
             py::arg("stop"),
             py::arg("num"),
             py::kw_only(),
             py::arg("grad") = false)
        .def("rand", &ryupy::Tensor::random_uniform,
             py::arg("shape"),
             py::kw_only(),
             py::arg("low") = 0.0f,
             py::arg("high") = 1.0f,
             py::arg("grad") = false)
        .def("randn", &ryupy::Tensor::random_normal,
             py::arg("shape"),
             py::kw_only(),
             py::arg("mean") = 0.0f,
             py::arg("std") = 1.0f,
             py::arg("grad") = false);

    auto nn = m.def_submodule("nn");

    using InitType = ryupy::nn::LinearLayer::InitType;
    py::enum_<InitType>(nn, "InitType")
        .value("XAVIER_UNIFORM", InitType::XAVIER_UNIFORM)
        .value("XAVIER_NORMAL", InitType::XAVIER_NORMAL)
        .value("KAIMING_UNIFORM", InitType::KAIMING_UNIFORM)
        .value("KAIMING_NORMAL", InitType::KAIMING_NORMAL)
        .export_values();

    py::class_<ryupy::nn::LinearLayer>(nn, "Linear")
        .def(py::init<int, int, InitType>())
        .def("forward", &ryupy::nn::LinearLayer::forward)
        .def_readwrite("weight", &ryupy::nn::LinearLayer::weight)
        .def_readwrite("bias", &ryupy::nn::LinearLayer::bias);
}